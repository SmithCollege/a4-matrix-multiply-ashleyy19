#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


// CUDA kernel for matrix multiplication
__global__ void matrix_multiply_gpu(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float value = 0.0;
        for (int k = 0; k < N; k++) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

int main() {
    // Matrix dimensions
    int N = 1024; // Example size, can vary
    size_t bytes = N * N * sizeof(float);

    // Host memory allocation
    float* h_A = (float*)malloc(bytes);
    float* h_B = (float*)malloc(bytes);
    float* h_C = (float*)malloc(bytes);

    // Initialize host matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f; 
        h_B[i] = 1.0f;
    }

    // Device memory allocation
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Start the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch the kernel
    matrix_multiply_gpu<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Stop the timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("GPU Matrix Multiply Time: %f ms\n", milliseconds);

    // Copy result back to host
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

cleanup:
    // Free host and device memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

