#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void tiled_matrix_multiply(float* A, float* B, float* C, int N) {
    __shared__ float Asub[16][16];
    __shared__ float Bsub[16][16];

    int tx = threadIdx.x, ty = threadIdx.y;
    int row = blockIdx.y * blockDim.y + ty;
    int col = blockIdx.x * blockDim.x + tx;

    float value = 0.0;

    for (int t = 0; t < (N + 16 - 1) / 16; t++) {
        // Load tiles into shared memory
        if (row < N && t * 16 + tx < N)
            Asub[ty][tx] = A[row * N + t * 16 + tx];
        else
            Asub[ty][tx] = 0.0;

        if (col < N && t * 16 + ty < N)
            Bsub[ty][tx] = B[(t * 16 + ty) * N + col];
        else
            Bsub[ty][tx] = 0.0;

        __syncthreads(); // Synchronize threads to ensure tiles are fully loaded

        // Perform multiplication for the current tile
        for (int k = 0; k < 16; k++) {
            value += Asub[ty][k] * Bsub[k][tx];
        }

        __syncthreads(); // Synchronize before loading the next tile
    }

    // Write result to global memory
    if (row < N && col < N) {
        C[row * N + col] = value;
    }
}

int main() {
    // Matrix dimensions
    int N = 1024; // Matrix size (N x N)
    size_t bytes = N * N * sizeof(float);

    // Host memory allocation
    float* h_A = (float*)malloc(bytes);
    float* h_B = (float*)malloc(bytes);
    float* h_C = (float*)malloc(bytes);

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
    }

    // Device memory allocation
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threadsPerBlock(16, 16); // Tile size is 16x16
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Start the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch the kernel
    tiled_matrix_multiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Stop the timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Tiled GPU Matrix Multiply Time: %f ms\n", milliseconds);

    // Copy result back to host
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (h_C[i * N + j] != N) {
                printf("Verification failed at (%d, %d): %f\n", i, j, h_C[i * N + j]);
                goto cleanup;
            }
        }
    }
    printf("Result verification passed.\n");

cleanup:
    // Free host and device memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
